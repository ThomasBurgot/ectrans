#define cufftSafeCall(err) __cufftSafeCall(err, __FILE__, __LINE__)

#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "stdio.h"
    static const char *_cudaGetErrorEnum(hipfftResult error)
    {
    switch (error)
    {
    case HIPFFT_SUCCESS:
    return "CUFFT_SUCCESS";

    case HIPFFT_INVALID_PLAN:
    return "CUFFT_INVALID_PLAN";

    case HIPFFT_ALLOC_FAILED:
    return "CUFFT_ALLOC_FAILED";

    case HIPFFT_INVALID_TYPE:
    return "CUFFT_INVALID_TYPE";

    case HIPFFT_INVALID_VALUE:
    return "CUFFT_INVALID_VALUE";

    case HIPFFT_INTERNAL_ERROR:
    return "CUFFT_INTERNAL_ERROR";

    case HIPFFT_EXEC_FAILED:
    return "CUFFT_EXEC_FAILED";

    case HIPFFT_SETUP_FAILED:
    return "CUFFT_SETUP_FAILED";

    case HIPFFT_INVALID_SIZE:
    return "CUFFT_INVALID_SIZE";

    case HIPFFT_UNALIGNED_DATA:
    return "CUFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
    }

    inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
    {
    if( HIPFFT_SUCCESS != err) {
    fprintf(stderr, "CUFFT error at 1\n");
    fprintf(stderr, "CUFFT error in file '%s'\n",__FILE__);
    fprintf(stderr, "CUFFT error at 2\n");
    /*fprintf(stderr, "CUFFT error line '%s'\n",__LINE__);*/
    fprintf(stderr, "CUFFT error at 3\n");
    /*fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
    _cudaGetErrorEnum(err)); \*/
    fprintf(stderr, "CUFFT error %d: %s\nterminating!\n",err,_cudaGetErrorEnum(err)); \
    hipDeviceReset(); return; \
    } /*else {
		fprintf(stderr, "CUFFT call at %s, %i returned code %s\n",file,line,_cudaGetErrorEnum(err));
	}*/
    }

extern "C"
void
#ifdef TRANS_SINGLE
execute_plan_fftc_(cufftHandle *PLANp, int *ISIGNp, cufftComplex *data_in, cufftComplex *data_out)
#else
execute_plan_fftc_(hipfftHandle *PLANp, int *ISIGNp, hipfftDoubleComplex *data_in, hipfftDoubleComplex *data_out)
#endif
{
hipfftHandle plan = *PLANp;
int ISIGN = *ISIGNp;

/*if (cudaDeviceSynchronize() != cudaSuccess){
	fprintf(stderr, "Cuda error: Failed to synchronize\n");
	return;	
}*/

if( ISIGN== -1 ){
  #ifdef TRANS_SINGLE
  cufftSafeCall(cufftExecR2C(plan, (cufftReal*)data_in, data_out));
  #else
  cufftSafeCall(hipfftExecD2Z(plan, (hipfftDoubleReal*)data_in, data_out));
  #endif
}
else if( ISIGN== 1){
  #ifdef TRANS_SINGLE
  cufftSafeCall(cufftExecC2R(plan, data_in, (cufftReal*)data_out));
  #else
  cufftSafeCall(hipfftExecZ2D(plan, data_in, (hipfftDoubleReal*)data_out));
  #endif
}
else {
  abort();
}

// cudaDeviceSynchronize();

//if (cudaDeviceSynchronize() != cudaSuccess){
//	fprintf(stderr, "Cuda error: Failed to synchronize\n");
//	return;	
//}


}



extern "C"
void
#ifdef TRANS_SINGLE
execute_plan_fftc_inplace_ (cufftHandle * PLANp, int * ISIGNp, cufftComplex * data)
#else
execute_plan_fftc_inplace_ (hipfftHandle * PLANp, int * ISIGNp, hipfftDoubleComplex * data)
#endif
{
  hipfftHandle plan = *PLANp;
  int ISIGN = *ISIGNp;
  
/*
	fprintf(stderr,"%s, %i : executing plan %i\n",__FILE__,__LINE__,*PLANp);
	
  fprintf(stderr,"%s, %i : cudaDeviceSynchronize returns code %i\n",__FILE__,__LINE__,cudaDeviceSynchronize());
*/
  
/*if (cudaDeviceSynchronize() != cudaSuccess){
  	fprintf(stderr, "Cuda error: Failed to synchronize\n");
  	return;	
}*/
  
  if (ISIGN== -1)
    {
#ifdef TRANS_SINGLE
    cufftSafeCall(cufftExecR2C(plan, (cufftReal*)data, data));
#else
//fprintf(stderr,"%s, %i : cudaDeviceSynchronize returns code %i\n",__FILE__,__LINE__,cudaDeviceSynchronize());
    cufftSafeCall(hipfftExecD2Z(plan, (hipfftDoubleReal*)data, data));
//fprintf(stderr,"%s, %i : cudaDeviceSynchronize returns code %i\n",__FILE__,__LINE__,cudaDeviceSynchronize());
	
#endif
    }
  else if (ISIGN== 1)
    {
#ifdef TRANS_SINGLE
    cufftSafeCall(cufftExecC2R(plan, data, (cufftReal*)data));
#else
    cufftSafeCall(hipfftExecZ2D(plan, data, (hipfftDoubleReal*)data));
#endif
    }
  else 
    {
      abort();
    }

// cudaDeviceSynchronize();

//if (cudaDeviceSynchronize() != cudaSuccess){
//	fprintf(stderr, "Cuda error: Failed to synchronize\n");
//	return;	
//}


}

